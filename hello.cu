#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>



__global__ void test()
{
#if __CUDA_ARCH__ >= 200
    printf("Hi Cuda World");
#endif
}

int main( int argc, char** argv )
{
    test<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
